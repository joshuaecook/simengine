// GPU execution kernel that runs each model instance for a number of iterations or until the buffer fills
__GLOBAL__ void exec_kernel_gpu(solver_props *props, int resuming){
  const unsigned int modelid = blockIdx.x * blockDim.x + threadIdx.x;
  
  unsigned int i, num_iterations = 0;
  CDATAFORMAT min_time;
  unsigned int ready_outputs[NUM_ITERATORS];
  int inputs_available = 1;

  for (i=0; i<NUM_ITERATORS; i++) {
    if(!resuming){
      dirty_states[i * PARALLEL_MODELS + modelid] = 0;
    }
    ready_outputs[i] = 0;
  }

  if (modelid >= props->num_models) {
    return;
  }

  if (!model_running(props, modelid)) {
    return;
  }

  // Initialize output buffer to store output data
  init_output_buffer(gpu_ob, modelid);

  // Update occurs before the first iteration
  for(i=0;i<NUM_ITERATORS;i++){
    if(props[i].running[modelid] && !resuming) {
      dirty_states[i * PARALLEL_MODELS + modelid] = 0 == update(&props[i], modelid);
    }	  
  }
  for(i=0;i<NUM_ITERATORS;i++){
    if (dirty_states[i * PARALLEL_MODELS + modelid] && !resuming) {
      solver_writeback(&props[i], modelid);
      dirty_states[i * PARALLEL_MODELS + modelid] = 0;
    }
  }

  min_time = find_min_time(props, modelid);

  // Run up to MAX_ITERATIONS iterations until the output buffer is full or the simulation is complete
  while (1) {
    // Cannot continue if all the simulation is complete
    if (!model_running(props, modelid)) {
      gpu_ob->finished[modelid] = 1;
      break;
    }

    // Stop if the maximum number of iterations has been executed
    if (num_iterations++ >= MAX_ITERATIONS) {
      break;
    }

    // Cannot continue if the output buffer is full
    if (gpu_ob->full[modelid]) {
      break;
    }

    // Preprocess phase: x[t] = f(x[t])
    for(i=0;i<NUM_ITERATORS;i++){
      if(props[i].running[modelid] && props[i].time[modelid] == min_time){
	dirty_states[i * PARALLEL_MODELS + modelid] = 0 == pre_process(&props[i], modelid);
      }
    }
    for(i=0;i<NUM_ITERATORS;i++){
      if (dirty_states[i * PARALLEL_MODELS + modelid] && props[i].time[modelid] == min_time) {
	solver_writeback(&props[i], modelid);
	dirty_states[i * PARALLEL_MODELS + modelid] = 0;
      }
    }

    // Main solver evaluation phase, including inprocess: x[t+dt] = f(x[t])
    for(i=0;i<NUM_ITERATORS;i++){
      if(props[i].running[modelid] && props[i].time[modelid] == min_time){
	// TODO check return status
	solver_eval(&props[i], modelid);
	// Now next_time == time + dt
	dirty_states[i * PARALLEL_MODELS + modelid] = 1;
	ready_outputs[i] = 1;
	// Run any in-process algebraic evaluations
	in_process(&props[i], modelid);
      }
    }

    // Find the nearest next_time and catch up
    min_time = find_min_time(props, modelid);

#if NUM_SAMPLED_INPUTS > 0
    // Advance any sampled inputs
    for (i=NUM_CONSTANT_INPUTS; i<NUM_CONSTANT_INPUTS + NUM_SAMPLED_INPUTS; i++) {
      sampled_input_t *input = &sampled_inputs[STRUCT_IDX * NUM_SAMPLED_INPUTS + SAMPLED_INPUT_ID(i)];
      inputs_available &= advance_sampled_input(input, min_time, props->modelid_offset, modelid);
    }
#endif

    // Buffer any available outputs
    for(i=0;i<NUM_ITERATORS;i++){
      if (ready_outputs[i]) {
#if NUM_OUTPUTS > 0
	buffer_outputs(&props[i], modelid);
#endif
	ready_outputs[i] = 0;
      }
      if (dirty_states[i * PARALLEL_MODELS + modelid] && props[i].next_time[modelid] == min_time) {
	solver_writeback(&props[i], modelid);
	dirty_states[i * PARALLEL_MODELS + modelid] = 0;
      }
    }

    // Cannot continue if a no inputs data are buffered
    if(!inputs_available) {
      if (1 == num_iterations) {
	  gpu_ob->finished[modelid] = 1;
      }
      break;
    }

    // Update and postprocess phase: x[t+dt] = f(x[t+dt])
    for(i=0;i<NUM_ITERATORS;i++){
      if(props[i].running[modelid] && props[i].next_time[modelid] == min_time) {
	dirty_states[i * PARALLEL_MODELS + modelid] = 0 == update(&props[i], modelid);
      }	  
      if(props[i].running[modelid] && props[i].next_time[modelid] == min_time) {
	dirty_states[i * PARALLEL_MODELS + modelid] |= 0 == post_process(&props[i], modelid);
      }
    }

    // Advance the iterator.
    for(i=0;i<NUM_ITERATORS;i++){
      if(props[i].running[modelid] && props[i].next_time[modelid] == min_time) {
	// Now time == next_time
	solver_advance(&props[i], modelid);
      }
    }
    for(i=0;i<NUM_ITERATORS;i++){
      if (dirty_states[i * PARALLEL_MODELS + modelid] && props[i].next_time[modelid] == min_time) {
	solver_writeback(&props[i], modelid);
	dirty_states[i * PARALLEL_MODELS + modelid] = 0;
      }
    }

    // Cannot continue if the output buffer is full
    if (gpu_ob->full[modelid]) {
      break;
    }

    // Capture outputs for final iteration
    for(i=0;i<NUM_ITERATORS;i++){
      if (props[i].last_iteration[modelid]) {
	props[i].last_iteration[modelid] = 0;

	pre_process(&props[i], modelid);
	model_flows(props[i].time[modelid], props[i].model_states, props[i].next_states, &props[i], 1, modelid);
	in_process(&props[i], modelid);

#if NUM_OUTPUTS > 0
	buffer_outputs(&props[i], modelid);
#endif
      }
    }
  }
}
