
#include <hip/hip_runtime.h>
#if defined(TARGET_GPU)
// GPU execution kernel that runs each model instance for a number of iterations or until the buffer fills
__GLOBAL__ void exec_kernel_gpu(INTEGRATION_MEM *mem, uint ob_id){
  const unsigned int modelid = blockIdx.x * blockDim.x + threadIdx.x;
  
  unsigned int num_iterations;
	     
  if (modelid < NUM_MODELS) {
    output_buffer *ob = (output_buffer *)mem->props->ob + ob_id;


    // Initialize output buffer to store output data
    init_output_buffer(ob, modelid);
    
    // Run up to MAX_ITERATIONS for each model
    for(num_iterations = 0; num_iterations < MAX_ITERATIONS; num_iterations++){
      // Check if simulation finished previously
      if(ob->finished[modelid] || ob->full[modelid]){
	// (threads are launched in batches on the GPU and not all will complete at the
	// same time with variable timestep solvers)
	break;
      }
      // Check if the simulation just finished (or if there are no states)
      if(!mem->props->running[modelid] || mem->props->statesize == 0){
	mem->props->running[modelid] = 0;
	ob->finished[modelid] = 1;
#if NUM_OUTPUTS > 0
	// Log output values for final timestep
	// Run the model flows to ensure that all intermediates are computed, mem->k1 is borrowed from the solver as scratch for ignored dydt values
	model_flows(mem->props->time[modelid], mem->props->model_states, mem->k1, mem->props->inputs, mem->props->outputs, 1, modelid);
	// Buffer the last values
	buffer_outputs(mem->props->time[modelid],((output_data*)mem->props->outputs), ob, modelid);
#endif
	break;
      }
      
      CDATAFORMAT prev_time = mem->props->time[modelid];

      // Execute solver for one timestep
      SOLVER(INTEGRATION_METHOD, eval, TARGET, SIMENGINE_STORAGE, mem, modelid);

#if NUM_OUTPUTS > 0
      // Store a set of outputs only if the sovler made a step
      if (mem->props->time[modelid] > prev_time) {
	buffer_outputs(prev_time, (output_data*)mem->props->outputs, ob, modelid);
      }
#endif
    }
  }
}
#endif
