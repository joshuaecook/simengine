int exec_parallel_gpu(solver_props *props, const char *outputs_dirname, double *progress, int resuming){
  unsigned int i;
  unsigned int inputid;
  unsigned int modelid;
  unsigned int iterid = NUM_ITERATORS - 1;
  unsigned int num_gpu_threads;
  unsigned int num_gpu_blocks;
  unsigned int active_models;
  solver_props *device_props;
#if NUM_SAMPLED_INPUTS > 0
  sampled_input_t tmp_sampled_inputs[STRUCT_SIZE * NUM_SAMPLED_INPUTS];
#endif

  num_gpu_threads = GPU_BLOCK_SIZE < props->num_models ? GPU_BLOCK_SIZE : props->num_models;
  num_gpu_blocks = (props->num_models + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE;

  // Initialize all iterators to running
  active_models = 1;
  for(modelid = 0; modelid < props->num_models; modelid++){
    for(i=0;i<NUM_ITERATORS;i++){
      props[i].running[modelid] = 1;
    }
  }
  
  // Initialize GPU device memory for all solvers (returns pointer to device memory)
  device_props = gpu_init_props(props);

  while(active_models){
    // Execute models on the GPU
    
    exec_kernel_gpu<<<num_gpu_blocks, num_gpu_threads>>>(device_props, resuming);
    resuming = 1;
    // Copy data back to the host
    cutilSafeCall(hipMemcpyFromSymbol(&global_ob[global_ob_idx[0]], HIP_SYMBOL(gpu_ob), sizeof(output_buffer), 0, hipMemcpyDeviceToHost));
    // Grab time from the last iterator (to be sure to skip an always iterator if it is present)
    cutilSafeCall(hipMemcpy(props[iterid].time, props[iterid].gpu.time, props[iterid].num_models * sizeof(CDATAFORMAT), hipMemcpyDeviceToHost));
#if NUM_SAMPLED_INPUTS > 0
    cutilSafeCall(hipMemcpyFromSymbol(tmp_sampled_inputs, HIP_SYMBOL(sampled_inputs), STRUCT_SIZE * NUM_SAMPLED_INPUTS * sizeof(sampled_input_t), 0, hipMemcpyDeviceToHost));
#endif

    active_models = 0;
    // Copy data to external api interface
    for(modelid = 0; modelid < props->num_models; modelid++){
      active_models |= !global_ob[global_ob_idx[modelid]].finished[modelid];
      progress[modelid] = (props[iterid].time[modelid] - props[iterid].starttime) / (props[iterid].stoptime - props[iterid].starttime);
#if NUM_SAMPLED_INPUTS > 0
      if (!global_ob[global_ob_idx[modelid]].finished[modelid]) {
	for (inputid = NUM_CONSTANT_INPUTS; inputid < NUM_CONSTANT_INPUTS + NUM_SAMPLED_INPUTS; inputid++) {
	  sampled_input_t *input = &tmp_sampled_inputs[STRUCT_IDX * NUM_INPUTS + SAMPLED_INPUT_ID(inputid)];
	  if (input->idx[ARRAY_IDX] >= input->buffered_size[ARRAY_IDX]) {
	    read_sampled_input(input, props->time[ARRAY_IDX], outputs_dirname, inputid, props->num_models, props->modelid_offset, modelid);
	    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(sampled_inputs), input, sizeof(sampled_input_t), SAMPLED_INPUT_ID(inputid) * sizeof(sampled_input_t), hipMemcpyHostToDevice));
	  }
	}
      }
#endif
      if(0 != log_outputs(outputs_dirname, props->modelid_offset, modelid)) return ERRMEM;
    }
  }

  // Copy any remaining data back from GPU
  gpu_finalize_props(props);

  return SUCCESS;
}
