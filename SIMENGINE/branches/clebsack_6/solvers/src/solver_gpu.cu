#include"solvers.h"
#include<cutil_inline.h>

void GPU_ENTRY(init, SIMENGINE_STORAGE){
  // FIXME Add more checking of capabilities and devices available!
  hipSetDevice(cutGetMaxGflopsDeviceId());
}

void GPU_ENTRY(exit, GPU, SIMENGINE_STORAGE){
  hipDeviceReset();
}

// Takes a solver_props pointer on the CPU and returns a pointer to a mirrored structure on the GPU
solver_props *GPU_ENTRY(init_props, SIMENGINE_STORAGE, solver_props *props){
  // Local temp
  solver_props tprops;

  // GPU datastructures
  solver_props *dprops;

  // Copy the properties to local temporary
  memcpy(&tprops, props, sizeof(solver_props));

  // Allocate GPU space for props and all pointer fields of props
  cutilSafeCall(hipMalloc((void**)&dprops, sizeof(solver_props)));
  cutilSafeCall(hipMalloc((void**)&tprops.time, props->num_models*sizeof(CDATAFORMAT)));
  cutilSafeCall(hipMalloc((void**)&tprops.model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT)));
  cutilSafeCall(hipMalloc((void**)&tprops.inputs, props->inputsize*sizeof(CDATAFORMAT)));
  cutilSafeCall(hipMalloc((void**)&tprops.ob, tprops.ob_size));

  // Copy props to GPU
  cutilSafeCall(hipMemcpy(dprops, &tprops, sizeof(solver_props), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.time, props->time, props->num_models*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.model_states, props->model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.inputs, props->inputs, props->num_models*props->inputsize, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.ob, props->ob, tprops.ob_size, hipMemcpyHostToDevice));

  props->ob = tprops.ob;
  return dprops;
}

// Frees a GPU solver props structure
void GPU_ENTRY(free_props, SIMENGINE_STORAGE, solver_props *props){
  solver_props tprops;

  cutilSafeCall(hipMemcpy(&tprops, props, sizeof(solver_props), hipMemcpyDeviceToHost));

  cutilSafeCall(hipFree(tprops.time));
  cutilSafeCall(hipFree(tprops.model_states));
  cutilSafeCall(hipFree(tprops.inputs));
  cutilSafeCall(hipFree(tprops.ob));
  cutilSafeCall(hipFree(props));
}
