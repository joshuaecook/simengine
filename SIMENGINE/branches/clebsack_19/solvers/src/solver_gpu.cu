#include"solvers.h"

void GPU_ENTRY(init, SIMENGINE_STORAGE){
  // FIXME Add more checking of capabilities and devices available!
  hipSetDevice(cutGetMaxGflopsDeviceId());
}

void GPU_ENTRY(exit, SIMENGINE_STORAGE){
  hipDeviceReset();
}

// Takes a solver_props pointer on the CPU and returns a pointer to a mirrored structure on the GPU
solver_props *GPU_ENTRY(init_props, SIMENGINE_STORAGE, solver_props *props){
  // Local temp
  solver_props tprops;

  // GPU datastructures
  solver_props *dprops;

  // Copy the properties to local temporary
  memcpy(&tprops, props, sizeof(solver_props));

  // Allocate GPU space for props and all pointer fields of props
  cutilSafeCall(hipMalloc((void**)&dprops, sizeof(solver_props)));
  cutilSafeCall(hipMalloc((void**)&tprops.time, props->num_models*sizeof(CDATAFORMAT)));
  cutilSafeCall(hipMalloc((void**)&tprops.model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT)));
  cutilSafeCall(hipMalloc((void**)&tprops.inputs, props->num_models*props->inputsize*sizeof(CDATAFORMAT)));
  cutilSafeCall(hipMalloc((void**)&tprops.ob, props->ob_size));
  cutilSafeCall(hipMalloc((void**)&tprops.outputs, props->num_models*props->outputsize*sizeof(CDATAFORMAT)));

  // Copy props to GPU
  cutilSafeCall(hipMemcpy(dprops, &tprops, sizeof(solver_props), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.time, props->time, props->num_models*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.model_states, props->model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.inputs, props->inputs, props->num_models*props->inputsize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.ob, props->ob, props->ob_size, hipMemcpyHostToDevice));

  // Store pointers to GPU memory for data we need to be able to retrieve
  props->gpu.ob = tprops.ob;
  props->gpu.time = tprops.time;
  props->gpu.model_states = tprops.model_states;
  return dprops;
}

// Frees a GPU solver props structure
void GPU_ENTRY(free_props, SIMENGINE_STORAGE, solver_props *props){
  solver_props tprops;

  cutilSafeCall(hipMemcpy(&tprops, props, sizeof(solver_props), hipMemcpyDeviceToHost));

  cutilSafeCall(hipFree(tprops.time));
  cutilSafeCall(hipFree(tprops.model_states));
  cutilSafeCall(hipFree(tprops.inputs));
  cutilSafeCall(hipFree(tprops.ob));
  cutilSafeCall(hipFree(tprops.outputs));
  cutilSafeCall(hipFree(props));
}
