
#include <hip/hip_runtime.h>
#ifdef TARGET_GPU

// Variables in global device memory. Do not refer to these directly in user code!
// gpu_init_solver_props() returns a pointer to the global solver properties.

// Needs to be copied host-to-device. May be __constant__?
__DEVICE__ solver_props gpu_solver_props[NUM_ITERATORS];

// Needs to be copied host-to-device and device-to-host. May be __shared__?
__DEVICE__ CDATAFORMAT gpu_time[NUM_MODELS * NUM_ITERATORS];

// Does not need to be copied. May be __shared__?
__DEVICE__ CDATAFORMAT gpu_next_time[NUM_MODELS * NUM_ITERATORS];

// Needs to be copied host-to-device and device-to-host.
__DEVICE__ CDATAFORMAT gpu_system_states[NUM_MODELS * NUM_STATES];

// Does not need to be copied?
__DEVICE__ CDATAFORMAT gpu_next_states[NUM_MODELS * NUM_STATES];

// Needs to be copied host-to-device.
__DEVICE__ CDATAFORMAT gpu_inputs[NUM_MODELS * NUM_INPUTS];

// Needs to be copied device-to-host? May be __shared__?
__DEVICE__ int gpu_running[NUM_MODELS * NUM_ITERATORS];

// Needs to be copied device-to-host after each bunch of iterations.
__DEVICE__ output_buffer gpu_ob;


void gpu_init (void) {
  // FIXME Add more checking of capabilities and devices available!
  cudaSetDevice(cutGetMaxGflopsDeviceId());
}

void gpu_exit (void) {
  cudaThreadExit();
}


// Given a pointer to an array of solver properties having NUM_ITERATORS length,
// initializes a mirrored set of properties in device global memory.
solver_props *gpu_init_props (solver_props *props) {
  // A temporary host duplicate of the solver properties which will be copied to device global memory.
  solver_props tmp_props[NUM_ITERATORS];
  memcpy(tmp_props, props, NUM_ITERATORS * sizeof(solver_props));

  // Reassigns pointers within the duplicate properties structures to locations in device global memory.
  unsigned int i, states_offset = 0;
  for (i = 0; i < NUM_ITERATORS; i++) {
    // Copies start time to device.
    cutilSafeCall(cudaMemcpyToSymbol(gpu_time, props[i].time, NUM_MODELS * sizeof(CDATAFORMAT), i * NUM_MODELS, cudaMemcpyHostToDevice));

    // Each iterator has its own area of memory, all of the equal sizes
    tmp_props[i].time = gpu_time + (i * NUM_MODELS);
    tmp_props[i].next_time = gpu_next_time + (i * NUM_MODELS);
    tmp_props[i].running = gpu_running + (i * NUM_MODELS);

    // The amount of memory varies for each iterator
    tmp_props[i].model_states = gpu_system_states + (states_offset * NUM_MODELS);
    tmp_props[i].next_states = gpu_next_states + (states_offset * NUM_MODELS);

    states_offset += props[i].statesize;

    // Every iterator shares the same memory
    tmp_props[i].system_states = gpu_system_states;
    tmp_props[i].outputs = NULL; // not needed?
    tmp_props[i].inputs = gpu_inputs;

    // Pointers to device global memory that the host needs
    props[i].gpu.time = tmp_props[i].time;
    props[i].gpu.model_states = tmp_props[i].model_states;
    props[i].gpu.ob = &gpu_ob;
  }

  // Copies initial states to device (and to next states on device).
  cutilSafeCall(cudaMemcpyToSymbol(gpu_system_states, props[0].system_states, NUM_MODELS * NUM_STATES * sizeof(CDATAFORMAT), 0, cudaMemcpyHostToDevice));
  cutilSafeCall(cudaMemcpyToSymbol(gpu_next_states, props[0].system_states, NUM_MODELS * NUM_STATES * sizeof(CDATAFORMAT), 0, cudaMemcpyHostToDevice));

  // Copies inputs to device.
  cutilSafeCall(cudaMemcpyToSymbol(gpu_inputs, props[0].inputs, NUM_MODELS * NUM_INPUTS * sizeof(CDATAFORMAT), 0, cudaMemcpyHostToDevice));

  // Copies properties to device.
  cutilSafeCall(cudaMemcpyToSymbol(gpu_solver_props, tmp_props, NUM_ITERATORS * sizeof(solver_props), 0, cudaMemcpyHostToDevice));

  return gpu_solver_props;
}

// Copies final times and states back to host main memory.
void gpu_finalize_props (solver_props *props) {
  unsigned int i, states_offset = 0;
  for (i = 0; i < NUM_ITERATORS; i++) {
    // Each iterator has its own area of memory
    cutilSafeCall(cudaMemcpyFromSymbol(props[i].time, gpu_time, NUM_MODELS * sizeof(CDATAFORMAT), i * NUM_MODELS, cudaMemcpyDeviceToHost));
  }
  
  cutilSafeCall(cudaMemcpyFromSymbol(props[0].system_states, gpu_system_states, NUM_MODELS * NUM_STATES * sizeof(CDATAFORMAT), 0, cudaMemcpyDeviceToHost));
}
#endif // #ifdef TARGET_GPU
