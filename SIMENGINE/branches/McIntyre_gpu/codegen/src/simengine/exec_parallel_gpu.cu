int exec_parallel_gpu(solver_props *props){
  unsigned int i;
  unsigned int modelid;
  Iterator iter;
  unsigned int num_gpu_threads;
  unsigned int num_gpu_blocks;
  solver_props *device_props;
  num_gpu_threads = GPU_BLOCK_SIZE < NUM_MODELS ? GPU_BLOCK_SIZE : NUM_MODELS;
  num_gpu_blocks = (NUM_MODELS + GPU_BLOCK_SIZE - 1) / GPU_BLOCK_SIZE;

  // Initialize all iterators to running
  for(modelid = 0; modelid < NUM_MODELS; modelid++){
    for(i=0;i<NUM_ITERATORS;i++){
      iter = ITERATORS[i];
      props[iter].running[modelid] = 1;
    }
  }

  // Initialize GPU device memory for all solvers (returns pointer to device memory)
  device_props = gpu_init_props(props);

  while(((output_buffer*)props->ob)->active_models){
    // Execute models on the GPU
    exec_kernel_gpu<<<num_gpu_blocks, num_gpu_threads>>>(device_props);
    // Copy data back to the host
    cutilSafeCall(hipMemcpy(props->ob, props->gpu.ob, props->ob_size, hipMemcpyDeviceToHost));

    // Copy data in parallel to external api interface
    for(modelid = 0; modelid < props->num_models; modelid++){
      if(0 != log_outputs((output_buffer*)props->ob, outputs, modelid))
	return ERRMEM;
    }
  }

  // Copy any remaining data back from GPU
  gpu_finalize_props(props);

  return SUCCESS;
}
