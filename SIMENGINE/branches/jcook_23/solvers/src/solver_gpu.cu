#include"solvers.h"

void GPU_ENTRY(init, SIMENGINE_STORAGE){
  // FIXME Add more checking of capabilities and devices available!
  hipSetDevice(cutGetMaxGflopsDeviceId());
  hipSetDeviceFlags(hipDeviceMapHost);
}

void GPU_ENTRY(exit, SIMENGINE_STORAGE){
  hipDeviceReset();
}

// Takes a solver_props pointer on the CPU and returns a pointer to a mirrored structure on the GPU
solver_props *GPU_ENTRY(init_props, SIMENGINE_STORAGE, solver_props *props){
  // Local temp
  solver_props tprops;

  // GPU datastructures
  solver_props *dprops;

  void *ob;

  //  cutilSafeCall(hipMalloc((void**)&tprops.ob, props->ob_size));
  cutilSafeCall(hipHostAlloc(&ob, props->ob_size, hipHostMallocMapped | hipHostMallocPortable));
  memcpy(ob, props->ob, props->ob_size);
  props->gpu.ob = ob;

  

  // Copy the properties to local temporary
  memcpy(&tprops, props, sizeof(solver_props));

  // Allocate GPU space for props and all pointer fields of props
  cutilSafeCall(hipMalloc((void**)&dprops, sizeof(solver_props)));
  cutilSafeCall(hipMalloc((void**)&tprops.time, props->num_models*sizeof(CDATAFORMAT)));
  if (props->statesize) {
    cutilSafeCall(hipMalloc((void**)&tprops.model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT)));
  }
  else { tprops.model_states = 0; }

  if (props->inputsize) {
    cutilSafeCall(hipMalloc((void**)&tprops.inputs, props->num_models*props->inputsize*sizeof(CDATAFORMAT)));
  }
  else { tprops.inputs = 0; }

  if (props->outputsize) {
    cutilSafeCall(hipMalloc((void**)&tprops.outputs, props->num_models*props->outputsize*sizeof(CDATAFORMAT)));
  }
  else { tprops.outputs = 0; }

  if (0 != cutilSafeCall(hipHostGetDevicePointer(&tprops.ob, ob, 0))) {
      return 0;
      }

  // Copy props to GPU
  cutilSafeCall(hipMemcpy(dprops, &tprops, sizeof(solver_props), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.time, props->time, props->num_models*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.model_states, props->model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.inputs, props->inputs, props->num_models*props->inputsize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));


  // Pointers to GPU memory for data we need to be able to retrieve
  props->gpu.time = tprops.time;
  props->gpu.model_states = tprops.model_states;
  return dprops;
}

// Frees a GPU solver props structure
void GPU_ENTRY(free_props, SIMENGINE_STORAGE, solver_props *props){
  solver_props tprops;

  cutilSafeCall(hipMemcpy(&tprops, props, sizeof(solver_props), hipMemcpyDeviceToHost));

  if (tprops.time)
    { cutilSafeCall(hipFree(tprops.time)); }
  if (tprops.model_states)
    { cutilSafeCall(hipFree(tprops.model_states)); }
  if (tprops.inputs)
    { cutilSafeCall(hipFree(tprops.inputs)); }
  if (tprops.outputs)
    { cutilSafeCall(hipFree(tprops.outputs)); }
  if (tprops.running)
    { cutilSafeCall(hipFree(tprops.running)); }
  if (props)
    { cutilSafeCall(hipFree(props)); }
}
