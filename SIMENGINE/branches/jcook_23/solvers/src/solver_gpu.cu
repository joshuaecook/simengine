#include"solvers.h"

void GPU_ENTRY(init, SIMENGINE_STORAGE){
  // FIXME Add more checking of capabilities and devices available!
  hipSetDevice(cutGetMaxGflopsDeviceId());
  hipSetDeviceFlags(hipDeviceMapHost);
}

void GPU_ENTRY(exit, SIMENGINE_STORAGE){
  hipDeviceReset();
}

// Takes a solver_props pointer on the CPU and returns a pointer to a mirrored structure on the GPU
solver_props *GPU_ENTRY(init_props, SIMENGINE_STORAGE, solver_props *props){
  // Local temp
  solver_props tprops;

  // GPU datastructures
  solver_props *dprops;

  void *ob;

  // Copy the properties to local temporary
  memcpy(&tprops, props, sizeof(solver_props));

  // Allocate GPU space for props and all pointer fields of props
  PRINTF("Allocating %zd bytes on GPU for solver properties.\n", sizeof(solver_props));
  cutilSafeCall(hipMalloc((void**)&dprops, sizeof(solver_props)));
  cutilSafeCall(hipMalloc((void**)&tprops.time, props->num_models*sizeof(CDATAFORMAT)));
  if (props->statesize) {
    cutilSafeCall(hipMalloc((void**)&tprops.model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT)));
  }
  else { tprops.model_states = 0; }

  if (props->inputsize) {
    cutilSafeCall(hipMalloc((void**)&tprops.inputs, props->num_models*props->inputsize*sizeof(CDATAFORMAT)));
  }
  else { tprops.inputs = 0; }

  if (props->gpu.ob_mapped)
      {
      PRINTF("Initializing solver props with zero-copy output buffers.\n");
      cutilSafeCall(hipHostAlloc(&ob, props->ob_size, hipHostMallocMapped | hipHostMallocPortable));
      memset(ob, 0, props->ob_size);

      if (0 != cutilSafeCall(hipHostGetDevicePointer(&tprops.ob, ob, 0))) 
	  { return 0; }

      props->ob = ob;
      }
  else
      { 
      cutilSafeCall(hipMalloc((void**)&tprops.ob, props->ob_size));
      cutilSafeCall(hipMemset(tprops.ob, 0, props->ob_size));
      }

  if (props->outputsize) {
    cutilSafeCall(hipMalloc((void**)&tprops.outputs, props->num_models*props->outputsize*sizeof(CDATAFORMAT)));
  }
  else { tprops.outputs = 0; }

  cutilSafeCall(hipMalloc((void**)&tprops.running, props->num_models*sizeof(CDATAFORMAT)));



  // Pointers to GPU memory for data we need to be able to retrieve
  if (props->gpu.ob_mapped)
      { 
      tprops.gpu.ob = ob; 
      props->ob = ob;
      }
  else
      { 
      props->gpu.ob = tprops.ob; 
      }
  props->gpu.time = tprops.time;
  props->gpu.model_states = tprops.model_states;



  // Copy props to GPU
  cutilSafeCall(hipMemcpy(dprops, &tprops, sizeof(solver_props), hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpy(tprops.time, props->time, props->num_models*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));
  if (tprops.model_states)
      { cutilSafeCall(hipMemcpy(tprops.model_states, props->model_states, props->num_models*props->statesize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice)); }
  if (tprops.inputs)
      { cutilSafeCall(hipMemcpy(tprops.inputs, props->inputs, props->num_models*props->inputsize*sizeof(CDATAFORMAT), hipMemcpyHostToDevice)); }
  cutilSafeCall(hipMemcpy(tprops.running, props->running, props->num_models*sizeof(CDATAFORMAT), hipMemcpyHostToDevice));

  return dprops;
}

// Frees a GPU solver props structure
void GPU_ENTRY(free_props, SIMENGINE_STORAGE, solver_props *dprops){
  solver_props tprops;

  cutilSafeCall(hipMemcpy(&tprops, dprops, sizeof(solver_props), hipMemcpyDeviceToHost));

  if (tprops.gpu.ob_mapped)
      { cutilSafeCall(hipHostFree(tprops.gpu.ob)); }
  else
      { cutilSafeCall(hipFree(tprops.ob)); }

  if (tprops.time)
    { cutilSafeCall(hipFree(tprops.time)); }
  if (tprops.model_states)
    { cutilSafeCall(hipFree(tprops.model_states)); }
  if (tprops.inputs)
    { cutilSafeCall(hipFree(tprops.inputs)); }
  if (tprops.outputs)
    { cutilSafeCall(hipFree(tprops.outputs)); }
  if (tprops.running)
    { cutilSafeCall(hipFree(tprops.running)); }
  if (dprops)
    { cutilSafeCall(hipFree(dprops)); }
}
